#include "hip/hip_runtime.h"
/*
* This software is Copyright (c) 2011 Lukas Odzioba <lukas dot odzioba at gmail dot com> 
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
* Based on S3nf implementation http://openwall.info/wiki/john/MSCash2
*/

#include <stdio.h>
#include "../cuda_mscash2.h"
#include "cuda_common.cuh"
extern "C" void mscash2_gpu(mscash2_password *, mscash2_hash *, mscash2_salt *);

__constant__ mscash2_salt cuda_salt[1];

__host__ void md4_crypt(uint32_t * buffer, uint32_t * hash)
{
	uint32_t a;
	uint32_t b;
	uint32_t c;
	uint32_t d;

	a = 0xFFFFFFFF + buffer[0];
	a = (a << 3) | (a >> 29);
	d = INIT_D + (INIT_C ^ (a & 0x77777777)) + buffer[1];
	d = (d << 7) | (d >> 25);
	c = INIT_C + (INIT_B ^ (d & (a ^ INIT_B))) + buffer[2];
	c = (c << 11) | (c >> 21);
	b = INIT_B + (a ^ (c & (d ^ a))) + buffer[3];
	b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + buffer[4];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + buffer[5];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + buffer[6];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + buffer[7];
	b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + buffer[8];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + buffer[9];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + buffer[10];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + buffer[11];
	b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + buffer[12];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + buffer[13];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + buffer[14];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + buffer[15];
	b = (b << 19) | (b >> 13);

	a += ((b & (c | d)) | (c & d)) + buffer[0] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + buffer[4] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + buffer[8] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + buffer[12] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + buffer[1] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + buffer[5] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + buffer[9] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + buffer[13] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + buffer[2] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + buffer[6] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + buffer[10] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + buffer[14] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + buffer[3] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + buffer[7] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + buffer[11] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + buffer[15] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += (d ^ c ^ b) + buffer[0] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + buffer[8] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + buffer[4] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + buffer[12] + SQRT_3;
	b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + buffer[2] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + buffer[10] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + buffer[6] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + buffer[14] + SQRT_3;
	b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + buffer[1] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + buffer[9] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + buffer[5] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + buffer[13] + SQRT_3;
	b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + buffer[3] + SQRT_3;
	a = (a << 3) | (a >> 29);

	d += (c ^ b ^ a) + buffer[11] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + buffer[7] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + buffer[15] + SQRT_3;
	b = (b << 15) | (b >> 17);

	hash[0] = a + INIT_A;
	hash[1] = b + INIT_B;
	hash[2] = c + INIT_C;
	hash[3] = d + INIT_D;
}

__device__ __host__ void preproc(const uint8_t * key, uint32_t keylen,
    uint32_t * state, uint8_t var)
{
	int i;
	uint32_t W[16], temp;
	uint8_t ipad[64];

	for (i = 0; i < 64; i++)
		ipad[i] = var;

	for (i = 0; i < keylen; i++)
		ipad[i] = ipad[i] ^ key[i];

#pragma unroll 16
	for (i = 0; i < 16; i++)
		GET_WORD_32_BE(W[i], ipad, i * 4);
	
	uint32_t A = INIT_A;
	uint32_t B = INIT_B;
	uint32_t C = INIT_C;
	uint32_t D = INIT_D;
	uint32_t E = INIT_E;

	SHA1(A, B, C, D, E, W);

	state[0] = A + INIT_A;
	state[1] = B + INIT_B;
	state[2] = C + INIT_C;
	state[3] = D + INIT_D;
	state[4] = E + INIT_E;

}

__device__ void hmac_sha1(const uint8_t * key, uint32_t keylen,
    const uint8_t * input, uint32_t inputlen, uint8_t * output,
    uint32_t * ipad_state, uint32_t * opad_state)
{
	int i;
	uint32_t temp, W[16];
	uint32_t A, B, C, D, E;
	uint32_t state_A,state_B,state_C,state_D,state_E;
	uint8_t buf[64];
	uint32_t *src=(uint32_t*)buf;
	i=64/4;
	while(i--)
	  *src++=0;

	memcpy(buf, input, inputlen);
	buf[inputlen] = 0x80;
	PUT_WORD_32_BE((64 + inputlen) << 3, buf, 60);

	A = ipad_state[0];
	B = ipad_state[1];
	C = ipad_state[2];
	D = ipad_state[3];
	E = ipad_state[4];
	
	state_A=A;
	state_B=B;
	state_C=C;
	state_D=D;
	state_E=E;

	for (i = 0; i < 16; i++)
		GET_WORD_32_BE(W[i], buf, i * 4);

	SHA1(A, B, C, D, E, W);

	A += state_A;
	B += state_B;
	C += state_C;
	D += state_D;
	E += state_E;

	PUT_WORD_32_BE(A, buf, 0);
	PUT_WORD_32_BE(B, buf, 4);
	PUT_WORD_32_BE(C, buf, 8);
	PUT_WORD_32_BE(D, buf, 12);
	PUT_WORD_32_BE(E, buf, 16);

	buf[20] = 0x80;
	PUT_WORD_32_BE(0x2A0, buf, 60);

	A = opad_state[0];
	B = opad_state[1];
	C = opad_state[2];
	D = opad_state[3];
	E = opad_state[4];
	
	state_A=A;
	state_B=B;
	state_C=C;
	state_D=D;
	state_E=E;

	for (i = 0; i < 16; i++)
		GET_WORD_32_BE(W[i], buf, i * 4);

	SHA1(A, B, C, D, E, W);

	A += state_A;
	B += state_B;
	C += state_C;
	D += state_D;
	E += state_E;

	PUT_WORD_32_BE(A, output, 0);
	PUT_WORD_32_BE(B, output, 4);
	PUT_WORD_32_BE(C, output, 8);
	PUT_WORD_32_BE(D, output, 12);
	PUT_WORD_32_BE(E, output, 16);
}

__device__ void pbkdf2(const uint8_t * pass, const uint8_t * salt,
    int saltlen, uint8_t * out)
{
	uint8_t temp[SHA1_DIGEST_LENGTH];
	__shared__ uint8_t sbuf[THREADS][48];
	uint8_t* buf=sbuf[threadIdx.x];
	uint32_t ipad_state[5];
	uint32_t opad_state[5];
	int i, j;
	uint8_t tmp_out[16];

	i=48/4;
	uint32_t *src=(uint32_t*)buf;
	while(i--)
	  *src++=0;

	memcpy(buf, salt, saltlen);
	buf[saltlen + 3] = 0x01;

	preproc(pass, 16, ipad_state, 0x36);
	preproc(pass, 16, opad_state, 0x5c);

	hmac_sha1(pass, 16, buf, saltlen + 4, temp, ipad_state, opad_state);

	memcpy(tmp_out, temp, 20);

	for (i = 1; i < ITERATIONS; i++) {
		hmac_sha1(pass, 16, temp, SHA1_DIGEST_LENGTH, temp, ipad_state,
		    opad_state);

#pragma unroll 16
		for (j = 0; j < 16; j++)
			tmp_out[j] ^= temp[j];
	}
	memcpy(out, tmp_out, 20);
}


__global__ void pbkdf2_kernel(mscash2_password * inbuffer,
    mscash2_hash * outbuffer)
{
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint32_t username_len = (uint32_t) cuda_salt[0].length;

	pbkdf2((uint8_t *) inbuffer[idx].dcc_hash,
	    cuda_salt[0].unicode_salt, username_len << 1,
	    (uint8_t *) outbuffer[idx].v);

}

__host__ void mscash_cpu(mscash2_password * inbuffer, mscash2_hash * outbuffer,
    mscash2_salt * host_salt)
    {
      
      int i,idx = 0;
	uint32_t buffer[16];
	uint32_t nt_hash[16];
	uint8_t salt[64];
	memset(salt,0,64);
	uint8_t *username = host_salt->salt;
	uint32_t username_len = (uint32_t) host_salt->length;
	

	for (i = 0; i < (username_len >> 1) + 1; i++)
		((uint32_t *) salt)[i] =
		    username[2 * i] | (username[2 * i + 1] << 16);
	memcpy(host_salt->unicode_salt, salt, 64);



	for (idx = 0; idx < KEYS_PER_CRYPT; idx++) {

		uint8_t *password = inbuffer[idx].v;
		uint32_t password_len = inbuffer[idx].length;
		memset(nt_hash, 0, 64);
		memset(buffer, 0, 64);

		for (i = 0; i < password_len >> 1; i++)
			buffer[i] =
			    password[2 * i] | (password[2 * i + 1] << 16);

		if (password_len % 2 == 1)
			buffer[i] = password[password_len - 1] | 0x800000;
		else
			buffer[i] = 0x80;

		buffer[14] = password_len << 4;

		md4_crypt(buffer, nt_hash);

		memcpy((uint8_t *) nt_hash + 16, salt, username_len << 1);

		i = username_len + 8;

		if (username_len % 2 == 1)
			nt_hash[i >> 1] =
			    username[username_len - 1] | 0x800000;
		else
			nt_hash[i >> 1] = 0x80;

		nt_hash[14] = i << 4;

		md4_crypt(nt_hash, inbuffer[idx].dcc_hash);

	}

      
    }
__host__ void mscash2_gpu(mscash2_password * inbuffer, mscash2_hash * outbuffer,
    mscash2_salt * host_salt)
{
	
	mscash_cpu(inbuffer,outbuffer,host_salt);
	mscash2_password *cuda_inbuffer;
	mscash2_hash *cuda_outbuffer;
	size_t insize = sizeof(mscash2_password) * KEYS_PER_CRYPT;
	size_t outsize = sizeof(mscash2_hash) * KEYS_PER_CRYPT;
	
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_salt), host_salt,
		sizeof(mscash2_salt)));
	
	HANDLE_ERROR(hipMalloc(&cuda_inbuffer, insize));
	HANDLE_ERROR(hipMalloc(&cuda_outbuffer, outsize));

	HANDLE_ERROR(hipMemcpy(cuda_inbuffer, inbuffer, insize,
		hipMemcpyHostToDevice));

	pbkdf2_kernel <<< BLOCKS, THREADS >>> (cuda_inbuffer, cuda_outbuffer);

	HANDLE_ERROR(hipMemcpy(outbuffer, cuda_outbuffer, outsize,
		hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(cuda_inbuffer));
	HANDLE_ERROR(hipFree(cuda_outbuffer));

}
