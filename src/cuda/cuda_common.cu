/*
* This software is Copyright (c) 2011-2012 Lukas Odzioba <ukasz at openwall dot net>
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
*/
#ifndef _CUDA_COMMON_CU
#define _CUDA_COMMON_CU

#include <stdio.h>
#include <assert.h>

#include "cuda_common.cuh"

extern "C"
void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess) {
		fprintf(stderr, "%s in %s at line %d\n",
		    hipGetErrorString(err), file, line);
		if (err == hipErrorLaunchOutOfResources)
			fprintf(stderr, "Try decreasing THREADS in the corresponding hip/hip_runtime.h file. See doc/README-CUDA\n");
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))

extern "C"
static char *human_format(size_t size)
{
	char pref[] = { ' ', 'K', 'M', 'G', 'T', 'P', 'E', 'Z', 'Y' };
	int prefid = 0;
	static char ret[32];

	while (size > 1024) {
		size /= 1024;
		prefid++;
	}
	sprintf(ret, "%zd.%zd %cB", size, (size % 1024) / 100, pref[prefid]);
	return ret;
}

extern "C"
void cuda_device_list()
{
	int i, devices;
	hipError_t ret;

	ret = hipGetDeviceCount(&devices);
	if (ret == hipErrorNoDevice) {
		puts("Error: No CUDA-capable devices were detected by the installed CUDA driver.\n");
		exit(1);
	}
	if (ret == hipErrorInsufficientDriver) {
		puts("Error: The installed NVIDIA CUDA driver is older than the CUDA runtime library.\nThis is not a supported configuration. Update your display driver.\n");
		exit(1);
	}

	printf("%d CUDA devices found:\n", devices);
	for (i = 0; i < devices; i++) {
		hipDeviceProp_t devProp;
		int arch_cores_sm[] = { 1, 8, 32, 192 };

		hipGetDeviceProperties(&devProp, i);
		printf("\nCUDA Device #%d\n", i);
		printf("\tName:                          %s\n", devProp.name);
		printf("\tType:                          %s\n",
		       devProp.integrated ? "integrated" : "discrete");
		printf("\tCompute capability:            sm_%d%d\n",
		    devProp.major, devProp.minor);
		if (devProp.major <= 3)
		printf("\tNumber of stream processors:   %d (%d x %d)\n",
		       devProp.multiProcessorCount * arch_cores_sm[devProp.major],
		       devProp.multiProcessorCount, arch_cores_sm[devProp.major]);
		else
		printf("\tNumber of multiprocessors:     %d\n",
		    devProp.multiProcessorCount);
		printf("\tClock rate:                    %d Mhz\n",
		    devProp.clockRate / 1024);
		printf("\tTotal global memory:           %s%s\n",
		    human_format(devProp.totalGlobalMem + 200000000),
		    devProp.ECCEnabled ? " (ECC)" : "");
		printf("\tTotal shared memory per block: %s\n",
		    human_format(devProp.sharedMemPerBlock));
		printf("\tTotal constant memory:         %s\n",
		    human_format(devProp.totalConstMem));
		printf("\tL2 cache size                  %s\n",
		       human_format(devProp.l2CacheSize));
		printf("\tKernel execution timeout:      %s\n",
		    (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
		printf("\tConcurrent copy and execution: %s\n",
		    (devProp.deviceOverlap ? "Yes" : "No"));
		printf("\tConcurrent kernels support:    %s\n",
		    (devProp.concurrentKernels ? "Yes" : "No"));
		printf("\tWarp size:                     %d\n",
		    devProp.warpSize);
		printf("\tMax. GPRs/thread block         %d\n",
		    devProp.regsPerBlock);
		printf("\tMax. threads per block         %d\n",
		    devProp.maxThreadsPerBlock);
		printf("\tMax. resident threads per MP   %d\n",
		    devProp.maxThreadsPerMultiProcessor);
		puts("");
	}
}

extern "C"
void *cuda_pageLockedMalloc(void *w, unsigned int size)
{
	HANDLE_ERROR(hipHostAlloc((void **) &w, size, hipHostMallocDefault));
	return w;
}

extern "C"
void cuda_pageLockedFree(void *w)
{
	HANDLE_ERROR(hipHostFree(w));
}

/* cuda init must be called first to set device */
extern "C"
int cuda_getAsyncEngineCount()
{
	hipDeviceProp_t prop;
	int dev;
	hipGetDevice(&dev);
	hipGetDeviceProperties(&prop,dev);
	return prop.asyncEngineCount;
	//if CUDA<4.0 we should use prop.overlapSupported
}
#endif
